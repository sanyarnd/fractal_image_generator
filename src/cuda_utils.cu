#include "cuda_utils.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

namespace cuda_info {
hipDeviceProp_t devicePropertries(int device) {
  hipDeviceProp_t prop{};
  cudaCheckRet(hipGetDeviceProperties(&prop, device));
  return prop;
}

bool deviceIsPresent() {
  int num_devices{};
  hipGetDeviceCount(&num_devices);
  hipError_t code = hipGetLastError();
  return (code == hipSuccess) && (num_devices > 0);
}
} // namespace cuda_info

namespace cuda_gl {
hipGraphicsResource *registerBuffer(GLuint buf) {
  hipGraphicsResource *res = nullptr;
  cudaCheckRet(
      hipGraphicsGLRegisterBuffer(&res, buf, hipGraphicsRegisterFlagsNone));
  return res;
}

void unregisterBuffer(hipGraphicsResource *res) {
  cudaCheckRet(hipGraphicsUnregisterResource(res));
}

void *map(hipGraphicsResource *res) {
  if (cudaCheckRet(hipGraphicsMapResources(1, &res)) != hipSuccess) {
    return nullptr;
  }

  void *devPtr = nullptr;
  size_t size;
  if (cudaCheckRet(hipGraphicsResourceGetMappedPointer(&devPtr, &size, res)) != hipSuccess) {
    return nullptr;
  }
  return devPtr;
}

void unmap(hipGraphicsResource *res) {
  cudaCheckRet(hipGraphicsUnmapResources(1, (hipGraphicsResource **) &res));
}
} // namespace cuda_gl
