#include "cuda_utils.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

hipDeviceProp_t cudaDevicePropertries(int device) {
  hipDeviceProp_t prop{};
  cudaCheckRet(hipGetDeviceProperties(&prop, device));
  return prop;
}
